#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <iostream>
#include <map>
#include <string>
#include <fstream>
#include <functional>
#include <vector>

using std::string;
using std::cout;
using std::endl;
using std::map;
using std::ifstream;
using std::getline;
using std::vector;
using std::stoi;
using std::pair;
using std::mem_fun_ref;
using std::ofstream;

#define P2_FILE "P2"
#define MY_PATH "C:\\lena.ascii.pgm"
#define OUTPUT_PATH "updated_lena.pgm"

void remove_empty_strings(vector<string>& strings);
size_t split(const string& text, vector<string>& parameters, char separator);

struct coordinates;
struct pixel;

class Photo
{
public:
	Photo(string file_path);

	void filter_negative();
	void save_file(string file_path);
	~Photo();

private:
	vector<pixel*>* image_pixels_;
	int height_;
	int width_;
	int max_gray_value_;
	string image_comment_;

	// Loading image
	void load_image(const string file_path);
	void load_image_size(ifstream* input);
	void load_max_gray(ifstream* input);
	void add_pixel(int row_counter, int column_counter, vector<std::basic_string<char>>* image_row, int i) const;
	void load_pixels(ifstream* input, int row_counter, int column_counter, string current_line) const;

	// Pixel filter methods
	int Photo::negative(int value);
};

int main()
{
	string my_path = MY_PATH;
	auto photo = new Photo(my_path);
	photo->filter_negative();

	photo->save_file(OUTPUT_PATH);
	delete photo;
	return 0;
}

void remove_empty_strings(vector<string>& strings)
{
	vector<string>::iterator it = remove_if(strings.begin(), strings.end(), mem_fun_ref(&string::empty));
	// erase the removed elements
	strings.erase(it, strings.end());
}

size_t split(const string& text, vector<string>& parameters, const char separator)
{
	size_t pos = text.find(separator);
	size_t initialPos = 0;
	parameters.clear();

	// Decompose statement
	while (pos != string::npos)
	{
		parameters.push_back(text.substr(initialPos, pos - initialPos));
		initialPos = pos + 1;

		pos = text.find(separator, initialPos);
	}

	// Add the last one
	parameters.push_back(text.substr(initialPos, std::min(pos, text.size()) - initialPos + 1));

	remove_empty_strings(parameters);

	return parameters.size();
}

struct coordinates
{
	int x;
	int y;
};

struct pixel
{
	int value;
	coordinates* coordinates;
};


Photo::Photo(string file_path)
{
	image_pixels_ = new vector<pixel*>();
	load_image(file_path);
}

void Photo::filter_negative()
{
	for (int i = 0; i < image_pixels_->size(); i++)
	{
		image_pixels_->at(i)->value = negative(image_pixels_->at(i)->value);
	}
}

void Photo::save_file(string file_path)
{
	ofstream processed_file(file_path);

	processed_file << P2_FILE << "\n";
	processed_file << image_comment_ << "\n";
	processed_file << width_ << "  " << height_ << "\n";
	processed_file << max_gray_value_ << "\n";

	int current_row = 0;
	
	for (int i = 0; i < image_pixels_->size(); i++)
	{
		auto current_pixel = image_pixels_->at(i);

		if (current_pixel->coordinates->y != current_row)
		{
			current_row++;
			processed_file << "\n";
		} else
		{
			processed_file << " ";
		}
		processed_file << current_pixel->value << " ";
	}

	processed_file << "\n";
	processed_file.close();
}

void Photo::load_image_size(ifstream* input)
{
	string size_line;
	getline(*input, size_line);
	auto image_size = new vector<string>();
	split(size_line, *image_size, ' ');
	width_ = stoi(image_size->at(0));
	height_ = stoi(image_size->at(1));
	delete image_size;
}

void Photo::load_max_gray(ifstream* input)
{
	string max_gray;
	getline(*input, max_gray);
	max_gray_value_ = stoi(max_gray);
}

void Photo::add_pixel(int row_counter, int column_counter, vector<std::basic_string<char>>* image_row, int i) const
{
	const auto current_pixel = new pixel();
	const auto coords = new coordinates();

	coords->x = column_counter;
	coords->y = row_counter;

	current_pixel->coordinates = coords;
	current_pixel->value = stoi(image_row->at(i));
	image_pixels_->push_back(current_pixel);
}

void Photo::load_pixels(ifstream* input, int row_counter, int column_counter, string current_line) const
{
	while (getline(*input, current_line))
	{
		auto image_row = new vector<string>();
		split(current_line, *image_row, ' ');

		for (int i = 0; i < image_row->size(); i++)
		{
			add_pixel(row_counter, column_counter, image_row, i);
			column_counter++;
		}

		column_counter = 0;
		row_counter++;
		image_row->clear();
		delete image_row;
	}
}

int Photo::negative(int value)
{
	return max_gray_value_ - value;
}

void Photo::load_image(const string file_path)
{
	ifstream input(file_path);

	if (input.is_open())
	{
		string file_type;
		getline(input, file_type);
		if (file_type == P2_FILE)
		{
			getline(input, image_comment_);

			load_image_size(&input);
			load_max_gray(&input);
			int row_counter = 0;
			int column_counter = 0;

			string current_line;

			load_pixels(&input, row_counter, column_counter, current_line);
		}
	}
}


Photo::~Photo()
{
	delete image_pixels_;
}
